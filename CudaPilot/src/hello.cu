#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

// gloabl qualifier indicates to compiler that this function 
// runs on device rather than host
__global__ void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    y[i] = a * x[i] + y[i];
}

__global__ void myFirstKernel(void) {
}

void helloCuda() {
  // CUDA C trick: call device code from host code, i.e. "kernel call"
  myFirstKernel <<<1, 1>>> ();
  std::cout << "Hello CUDA" << std::endl;
}

int main(void)
{
  int N = 1 << 20;
  float *x, *y, *d_x, *d_y;
  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i] - 4.0f));
  printf("Max error: %f\n", maxError);

  helloCuda();

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
